#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/hip_runtime.h>
#include <>
//#include <hip/device_functions.h>
#include "hip/hip_runtime.h"


hipError_t decryptWithCuda(unsigned int *four_byte_val);

unsigned int hexadecimalToDecimal(char* hexVal) {
	unsigned short len = strlen(hexVal);
	unsigned int base = 1;
	unsigned int dec_val = 0;

	for (short i = len - 1; i >= 0; i--) {
		if (hexVal[i] >= '0' && hexVal[i] <= '9')
			dec_val += (hexVal[i] - 48) * base;
		else if (hexVal[i] >= 'a' && hexVal[i] <= 'f')
			dec_val += (hexVal[i] - 87) * base;

		base *= 16;
	}

	return dec_val;
}

char* decimalToASCII(unsigned int decimal) {
	unsigned short mask = (1 << 8) - 1;
  //char* string;
  //hipMallocManaged(&string, sizeof(char) * 5);
	unsigned short byte;
	short i;
	for (i = 3; i >= 0; i--) {
		byte = (decimal >> (i * 8)) & mask;
		if (byte < 32 || byte > 126) {
      //hipFree(string);
			return NULL;
		}
		//else
			//string[3 - i] = byte;
	}
	//string[4] = '\0';

	return NULL;
}

void decimalToHex(unsigned int decimal) {
	unsigned short mask = (1 << 4) - 1;
	unsigned short byte;
	for (int i = 7; i >= 0; i--) {
		byte = (decimal >> (i * 4)) & mask;
		if (byte < 10)
			byte += 48;
		else if (byte >= 10)
			byte += 87;

		printf("%c", (char)byte);
	}
}


__global__ void decrypt(unsigned int *four_byte_val) {
  unsigned int i = 512 * blockIdx.x + threadIdx.x;
  //unsigned int range = 4294967296;
  //unsigned int range2 = 16777216;
  char *temp;

  char string[185];
  string[184] = '\0';

  unsigned short mask = (1 << 8) - 1;
  char tmp_string[5];
  tmp_string[4] = '\0';

  unsigned int x;
  unsigned int xor;
  unsigned short ok, ok2, count, byte;
  short index, j, k;
  unsigned int left, right;
  left = i * 32768;
  right = (i + 1) * 32768 - 1;
  //printf("Launched thread %u with range (%u, %u)\n", i, left, right);
  for (x = left; x <= right; x++) {
    //printf("%u\n", x);
    count = 180;
    ok = 1;
    for (index = 46; index >= 1; index--) {
      xor = ((four_byte_val[index - 1] + x) % 0x100000000) ^ four_byte_val[index];
      ok2 = 1;
      for (k = 3; k >= 0; k--) {
        byte = (xor >> (k * 8)) & mask;
        if (byte < 32 || byte > 126) {
          ok2 = 0;
          break;
        }
        else {
          tmp_string[3 - k] = (char)byte;
        }
      }
      
      if (ok2 == 1) {
        for (j = count; j < count + 4; j++)
          string[j] = tmp_string[j % 4];
        count -= 4;
      }
      else {
        ok = 0;
        break;
      }
    }

    if (ok == 1) {
      //fprintf(fp, "%s\n", string);
      printf("%ld %s\n", x, string);
    }

    if (x == 4294967295)
      break;
    //printf("HERE %d\n", i);
  }
  //printf("EXITED %u\n", i);
}


int main() {
  time_t seconds = time(NULL);
	char *ciphertext = "5499fa991ee7d8da5df0b78b1cb0c18c10f09fc54bb7fdae7fcb95ace494fbae8f5d90a3c766fdd7b7399eccbf4af592f35c9dc2272be2a45e788697520febd8468c808c2e550ac92b4d28b74c16678933df0bec67a967780ffa0ce344cd2a9a2dc208dc35c26a9d658b0fd70d00648246c90cf828d72a794ea94be51bbc6995478505d37b1a6b8daf7408dbef7d7f9f76471cc6ef1076b46c911aa7e75a7ed389630c8df32b7fcb697c1e89091c30be736a4cbfe27339bb9a2a52a280";
  char *substr = (char *) malloc(sizeof(char) * 9);
  unsigned int *four_byte_val = (unsigned int *) malloc(sizeof(unsigned int) * 47);
  substr[8] = '\0';

  short i = 46;
  for (short index = strlen(ciphertext) - 10; index >= 0; index -= 8) {
    strncpy(substr, ciphertext + index, 8);
    four_byte_val[i--] = hexadecimalToDecimal(substr);
  }

  free(substr);
  for (size_t p = 0; p < 47; p++)
    printf("%u\n", four_byte_val[p]);

  free(four_byte_val);
  //FILE * fp = fopen("decrypt.txt", "w+");
  
  // Add vectors in parallel.
  hipError_t cudaStatus = decryptWithCuda(four_byte_val);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "decryptWithCuda failed!");
    return 1;
  }

  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;
  }

	//fclose(fp);
  printf("Program finished in %d seconds\n", (time(NULL) - seconds));

	return 0;
}

hipError_t decryptWithCuda(unsigned int *four_byte_val) {
  unsigned int* dev_four_byte_val = NULL;
  hipError_t cudaStatus;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // Allocate GPU buffers for three vectors (two input, one output).
  cudaStatus = hipMalloc((void**)&dev_four_byte_val, 47 * sizeof(unsigned int));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  // Copy input vectors from host memory to GPU buffers.
  cudaStatus = hipMemcpy(dev_four_byte_val, four_byte_val, 47 * sizeof(unsigned int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  // Launch a kernel on the GPU with one thread for each element.
  decrypt<<<512, 256>>>(dev_four_byte_val);

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "decryptKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    goto Error;
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching decryptKernel!\n", cudaStatus);
    goto Error;
  }

Error:
  hipFree(dev_four_byte_val);

  return cudaStatus;
}